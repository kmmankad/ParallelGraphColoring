#include "hip/hip_runtime.h"
/*
# * Parallel Graph Coloring: 
# * Author: Kartik Mankad 
# * Email: kmankad@ncsu.edu
# * Description: A parallel implementation of
#	         an MIS based graph coloring algorithm
# */
#include "MaxMinSetCUDA.h"

// Random Number generator init
__global__ void InitRandGen (int d_NumVertices, int RandSeed, hiprandState_t* RandStates){
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (thread_num < d_NumVertices) {
		// Initialization is much faster if sequence number and offset
		// are kept at zero, and instead a different seed is used.
		// See - https://devtalk.nvidia.com/default/topic/480586/hiprand-initialization-time/?offset=4
		hiprand_init(RandSeed+thread_num, /* sequence number */ 0, /* sequence offset */ 0, &RandStates[thread_num]);
	}

}

// Assign Random Numbers to each vertex.
// TODO: Check if merging the two RNG kernel calls helps perf
__global__ void AssignRand (int d_NumVertices, int d_NNZ, int* d_ColIdx, int* d_RowPtr, int* d_RandNums, hiprandState_t* RandStates){
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadID < d_NumVertices){
		d_RandNums[threadID] = hiprand(&RandStates[thread_num]) % d_NumVertices;
	}
}

// Actual Graph Coloring kernel
__global__ void ColorGraph(int d_NumVertices, int d_NNZ, int* d_ColIdx, int* d_RowPtr, int* d_ColorVector, int d_ColorVal, int* d_RandNums, bool* d_changed){
	int threadID = blockIdx.x * blockDim.x + threadIdx.x;
	// Set the default value of changed to false
	*d_changed = false;
	// So that we dont walk over the edge of the d_RowPtr array
	if (threadID <= d_NNZ) { 
		// if the vertex is not colored
		if (d_ColorVector[threadID] == NO_COLOR){
			// Iterate over its neighbors
			// and color it d_ColorVal if its the max
			bool VertexIsMax = true; 
			for (int CurrNodeOffset=d_RowPtr[threadID]; CurrNodeOffset<d_RowPtr[threadID+1] ; CurrNodeOffset++){
				// Ignore neighbors that are already colored 
				int NeighborColor = d_ColorVector[d_ColIdx[CurrNodeOffset]];
				if (NeighborColor == NO_COLOR){
					// Check if the vertex is the maximum of its neighbors
					if (d_RandNums[threadID] < d_RandNums[d_ColIdx[CurrNodeOffset]]){
						VertexIsMax = false;	
					}
				}
			}
			// If this vertex is the max, assign it d_ColorVal
			if (VertexIsMax == true){
				d_ColorVector[threadID] = d_ColorVal;
				*d_changed = true;
			}
		} // end if d_ColorVector[threadID] == NO_COLOR
	} // end if (threadID < d_NNZ)
}
